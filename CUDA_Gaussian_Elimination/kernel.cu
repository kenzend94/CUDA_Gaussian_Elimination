#include "hip/hip_runtime.h"
﻿//Kernel function that executes on the device 
#include "common.h" 
#include <hip/hip_runtime.h>

#include "hip/hip_runtime.h"
#include ""

__global__ void Kernel(float* a_d, float* b_d, int size)

{
    int idx = threadIdx.x;
    int idy = threadIdx.y;
    //int width = size ; 
    //int height = size ; 

    //Allocating memory in the share memory of the device 
    __shared__ float temp[16][16];

    //Copying the data to the shared memory 
    temp[idy][idx] = a_d[(idy * (size + 1)) + idx];

    for (int i = 1; i < size; i++)
    {
        if ((idy + i) < size) // NO Thread divergence here 
        {
            float var1 = (-1) * (temp[i - 1][i - 1] / temp[i + idy][i - 1]);
            temp[i + idy][idx] = temp[i - 1][idx] + ((var1) * (temp[i + idy][idx]));
        }
        __syncthreads(); //Synchronizing all threads before Next iterat ion 
    }

    b_d[idy * (size + 1) + idx] = temp[idy][idx];
}