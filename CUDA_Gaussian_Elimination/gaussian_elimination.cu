#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <fstream>
#include <chrono>
#include <string>
#include <vector>
#include <unistd.h>

using namespace std;


// define shouldPrint to print the matrix at each step
bool shouldPrint = true;

// define data_file to read the matrix from a file
const char *data_file = "data/data30.txt";

// Define a small threshold value
const double EPSILON = 1e-10;

// CUDA kernel for forward elimination
__global__ void forwardElimKernel(double *mat, int numvar, int k) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x + k + 1;
    if (idx < numvar) {
        double factor = mat[idx * (numvar + 1) + k] / mat[k * (numvar + 1) + k];
        for (int j = k; j <= numvar; j++) {
            mat[idx * (numvar + 1) + j] -= factor * mat[k * (numvar + 1) + j];
        }
    }
}

// Function to convert a string fraction to decimal
double fractionToDecimal(const string& frac) {
    istringstream iss(frac);
    double num, denom = 1;
    char slash;
    iss >> num >> slash >> denom;
    if (denom != 0) return num / denom;
    return num; // Handle non-fraction case
}

// Function to print the matrix with fractions
void printFractionMatrix(const vector<string>& mat, int numvar) {
    for (int i = 0; i < numvar; i++) {
        for (int j = 0; j <= numvar; j++) {
            cout << setw(10) << mat[i * (numvar + 1) + j] << " ";
        }
        cout << endl;
    }
    cout << endl;
}

double parseFraction(const string& frac) {
    size_t slashPos = frac.find('/');
    if (slashPos != string::npos) {
        double numerator = stod(frac.substr(0, slashPos));
        double denominator = stod(frac.substr(slashPos + 1));
        if (denominator != 0) return numerator / denominator;
    }
    return stod(frac); // Handle non-fraction case
}

// Function to find the greatest common divisor (GCD)
int gcd(int a, int b) {
    if (b == 0) {
        return a;
    }
    return gcd(b, a % b);
}

// Function to convert a decimal to a fraction
void decimalToFrac(double value, int &numerator, int &denominator) {
    // Check if the value is close to an integer
    double diff = fabs(value - round(value));
    if (diff < EPSILON) {
        numerator = static_cast<int>(round(value));
        denominator = 1;
        return; // Early return as the number is effectively an integer
    }

    const double precision = 1E-6; // Precision for the conversion
    double integral = floor(value);
    double frac = value - integral;
    const int max_denominator = 10000; // Limits the denominator size

    // Initialize denominator as 1
    int lower_n = 0;
    int lower_d = 1;
    int upper_n = 1;
    int upper_d = 1;

    while (lower_d <= max_denominator && upper_d <= max_denominator) {
        int middle_n = lower_n + upper_n;
        int middle_d = lower_d + upper_d;

        if (fabs(frac - (double)middle_n / middle_d) < precision) {
            if (middle_d > max_denominator) {
                if (lower_d > upper_d) {
                    lower_n = upper_n;
                    lower_d = upper_d;
                }
                break;
            }

            lower_n = upper_n = middle_n;
            lower_d = upper_d = middle_d;
        } else if (frac > (double)middle_n / middle_d) {
            lower_n = middle_n;
            lower_d = middle_d;
        } else {
            upper_n = middle_n;
            upper_d = middle_d;
        }
    }

    // Adjust fraction to combine with the integral part
    numerator = (int)integral * lower_d + lower_n;
    denominator = lower_d;

    // Reduce the fraction
    int commonDivisor = gcd(abs(numerator), denominator);
    numerator /= commonDivisor;
    denominator /= commonDivisor;
}

// Function to print the matrix
void printMatrix(double *mat, int numvar) {
    for (int i = 0; i < numvar; i++) {
        for (int j = 0; j <= numvar; j++) {
            // Convert each element to fraction
            int numerator, denominator;
            decimalToFrac(mat[i * (numvar + 1) + j], numerator, denominator);
            
            // Check if the value is smaller than EPSILON in absolute terms
            if (fabs(mat[i * (numvar + 1) + j]) < EPSILON) {
                cout << setw(10) << 0 << " ";
            } else {
                // Display as a fraction
                if (denominator == 1) { // Print as an integer if the denominator is 1
                    cout << setw(10) << numerator << " ";
                } else { // Otherwise, print as a fraction
                    cout << setw(10) << numerator << "/" << denominator << " ";
                }
            }
        }
        cout << endl;
    }
    cout << endl;
}


// Forward elimination function with an option to print matrix at each step
void forwardElim(double *mat, int numvar, bool printSteps) {
    double *d_mat;
    size_t size = numvar * (numvar + 1) * sizeof(double);
    hipMalloc(&d_mat, size);
    hipMemcpy(d_mat, mat, size, hipMemcpyHostToDevice);

    dim3 block(256);
    dim3 grid((numvar + block.x - 1) / block.x);

    for (int k = 0; k < numvar; k++) {
        forwardElimKernel<<<grid, block>>>(d_mat, numvar, k);
        hipDeviceSynchronize();

        if (printSteps) {
            // Copy back the matrix to the host to print
            hipMemcpy(mat, d_mat, size, hipMemcpyDeviceToHost);
            cout << "Matrix after step " << k << ":" << endl;
            printMatrix(mat, numvar);
        }
    }

    if (!printSteps) {
        hipMemcpy(mat, d_mat, size, hipMemcpyDeviceToHost);
    }
    hipFree(d_mat);
}


// 
void backSub(double *mat, int numvar) {
    double *x = new double[numvar];  // Dynamically allocate array for solution

    // Start the timer
    auto start = chrono::high_resolution_clock::now();

    for (int i = numvar - 1; i >= 0; i--) {
        x[i] = mat[i * (numvar + 1) + numvar];
        for (int j = i + 1; j < numvar; j++) {
            x[i] -= mat[i * (numvar + 1) + j] * x[j];
        }
        x[i] /= mat[i * (numvar + 1) + i];
    }
    // Stop the timer
    auto end = chrono::high_resolution_clock::now();

    // Calculate the duration
    auto duration = chrono::duration_cast<chrono::nanoseconds>(end - start);


    cout << "\nSolution for the system:\n";
    for (int i = 0; i < numvar; i++)
        cout << "X" << i << " = " << x[i] << endl;

    // Uncomment the following code to print the solution as fractions
    // for (int i = 0; i < numvar; i++) {
    //     int numerator, denominator;
    //     decimalToFrac(x[i], numerator, denominator);
    //     cout << "X" << i << " = " << numerator << "/" << denominator << endl;
    // }

    cout << "\nTime taken for back substitution: " << duration.count() << " nanoseconds" << endl;

    delete[] x; // Free the dynamically allocated memory
}


// Main function
int main() {
    ifstream file(data_file);
    if (!file.is_open()) {
        cerr << "Error opening file" << endl;
        return -1;
    }

    int numvar;
    file >> numvar;

    vector<string> matStr(numvar * (numvar + 1));
    string frac; // Temporary string to store fraction input

    // Read the matrix data as fractions or decimal numbers
    for (int i = 0; i < numvar; i++) {
        for (int j = 0; j <= numvar; j++) {
            file >> frac;
            matStr[i * (numvar + 1) + j] = frac;
        }
    }

    file.close();

    // Print the input matrix as fractions or decimal numbers
    cout << "Input Matrix:\n";
    printFractionMatrix(matStr, numvar);

    // Convert the string representations to decimal values for calculations
    double *mat = new double[numvar * (numvar + 1)];
    for (int i = 0; i < numvar * (numvar + 1); ++i) {
        mat[i] = fractionToDecimal(matStr[i]);
    }

    // Perform Gaussian elimination
    forwardElim(mat, numvar, shouldPrint);

    // Perform back substitution and print the results
    backSub(mat, numvar);

    // Free the dynamically allocated memory
    delete[] mat;

    return 0;
}
