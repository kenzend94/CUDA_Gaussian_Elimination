#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <fstream>

using namespace std;

#define N 3  // Number of unknowns

// Define a small threshold value
const double EPSILON = 1e-12;

// CUDA kernel for forward elimination
__global__ void forwardElimKernel(double *mat, int numvar, int k) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x + k + 1;
    if (idx < numvar) {
        double factor = mat[idx * (numvar + 1) + k] / mat[k * (numvar + 1) + k];
        for (int j = k; j <= numvar; j++) {
            mat[idx * (numvar + 1) + j] -= factor * mat[k * (numvar + 1) + j];
        }
    }
}

void makeRREF(double *mat, int numvar) {
    // Normalize diagonal elements to 1
    for (int i = 0; i < numvar; ++i) {
        double diagVal = mat[i * (numvar + 1) + i];
        if (diagVal != 0) { // Avoid division by zero
            for (int j = 0; j <= numvar; ++j) {
                mat[i * (numvar + 1) + j] /= diagVal;
            }
        }
    }

    // Zero out elements above the diagonal
    for (int i = numvar - 1; i >= 0; --i) {
        for (int j = i - 1; j >= 0; --j) {
            double factor = mat[j * (numvar + 1) + i];
            for (int k = 0; k <= numvar; ++k) {
                mat[j * (numvar + 1) + k] -= factor * mat[i * (numvar + 1) + k];
            }
        }
    }
}


// Function to print the matrix
void printMatrix(double *mat, int numvar) {
    for (int i = 0; i < numvar; i++) {
        for (int j = 0; j <= numvar; j++) {
            // Check if the value is smaller than EPSILON in absolute terms
            if (fabs(mat[i * (numvar + 1) + j]) < EPSILON) {
                cout << setw(10) << 0 << " ";
            } else {
                cout << setw(10) << mat[i * (numvar + 1) + j] << " ";
            }
        }
        cout << endl;
    }
    cout << endl;
}


// // Forward elimination on the GPU
// void forwardElim(double *mat, int numvar) {
//     double *d_mat;
//     size_t size = numvar * (numvar + 1) * sizeof(double);
//     cudaMalloc(&d_mat, size);
//     cudaMemcpy(d_mat, mat, size, cudaMemcpyHostToDevice);

//     dim3 block(256);
//     dim3 grid((numvar + block.x - 1) / block.x);

//     for (int k = 0; k < numvar; k++) {
//         forwardElimKernel<<<grid, block>>>(d_mat, numvar, k);
//         cudaDeviceSynchronize();
//     }

//     cudaMemcpy(mat, d_mat, size, cudaMemcpyDeviceToHost);
//     cudaFree(d_mat);
// }

// Print version will make it slower
// Modified forwardElim function to print matrix at each step with CUDA
void forwardElim(double *mat, int numvar) {
    double *d_mat;
    size_t size = numvar * (numvar + 1) * sizeof(double);
    hipMalloc(&d_mat, size);
    hipMemcpy(d_mat, mat, size, hipMemcpyHostToDevice);

    dim3 block(256);
    dim3 grid((numvar + block.x - 1) / block.x);

    for (int k = 0; k < numvar; k++) {
        forwardElimKernel<<<grid, block>>>(d_mat, numvar, k);
        hipDeviceSynchronize();

        // Copy back the matrix to the host to print
        hipMemcpy(mat, d_mat, size, hipMemcpyDeviceToHost);
        cout << "Matrix after step " << k << ":" << endl;
        printMatrix(mat, numvar);
    }

    hipMemcpy(mat, d_mat, size, hipMemcpyDeviceToHost);
    hipFree(d_mat);
}


// Function for back substitution
// void backSub(double mat[N][N+1]) {
//     double x[N];  // An array to store solution

//     for (int i = N-1; i >= 0; i--) {
//         x[i] = mat[i][N];
//         for (int j=i+1; j<N; j++) {
//             x[i] -= mat[i][j]*x[j];
//         }
//         x[i] = x[i]/mat[i][i];
//     }

//     cout << "\nSolution for the system:\n";
//     for (int i=0; i<N; i++)
//         cout << "X" << i << " = " << x[i] << endl;
// }

    void backSub(double *mat) {
        double x[N];  // An array to store solution

        for (int i = N-1; i >= 0; i--) {
            x[i] = mat[i * (N + 1) + N];
            for (int j=i+1; j<N; j++) {
                x[i] -= mat[i * (N + 1) + j] * x[j];
            }
            x[i] = x[i] / mat[i * (N + 1) + i];
        }

        cout << "\nSolution for the system:\n";
        for (int i=0; i<N; i++)
            cout << "X" << i << " = " << x[i] << endl;
    }

// Main function

int main() {
    ifstream file("data5.txt");
    if (!file.is_open()) {
        cerr << "Error opening file" << endl;
        return -1;
    }

    int numvar;
    file >> numvar;

    // Dynamically allocate memory for the matrix
    double *mat = new double[numvar * (numvar + 1)];

    // Read the matrix data
    for (int i = 0; i < numvar; i++) {
        for (int j = 0; j <= numvar; j++) {
            file >> mat[i * (numvar + 1) + j];
        }
    }

    // print matrix
    cout << "Input Matrix:\n";
    for (int i = 0; i < numvar; i++) {
        for (int j = 0; j <= numvar; j++) {
            cout << mat[i * (numvar + 1) + j] << " ";
        }
        cout << endl;
    }
    // why cout << endl; here?
    // cout << endl;


    file.close();

    // Call forward elimination and back substitution
    forwardElim(mat, numvar);
    // Function for back substitution
    backSub(mat);
    makeRREF(mat, numvar); // Transform to RREF

    cout << "Matrix in RREF:" << endl;
    printMatrix(mat, numvar);

    // Free the dynamically allocated memory
    delete[] mat;

    return 0;
}