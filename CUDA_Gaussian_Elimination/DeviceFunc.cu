#include "hip/hip_runtime.h"
//Assigning memory on device and defining Thread Block size 
// Call to the Kernel(function) that will run on the GPU 

#include<hip/hip_runtime.h> 
#include<stdio.h> 
#include "common.h" 
#include "hip/hip_runtime.h"
#include ""

__global__ void Kernel(float*, float*, int);

void DeviceFunc(float* temp_h, int numvar, float* temp1_h)
{
    float* a_d, * b_d;

    //Memory allocation on the device 
    hipMalloc(&a_d, sizeof(float) * (numvar) * (numvar + 1));
    hipMalloc(&b_d, sizeof(float) * (numvar) * (numvar + 1));

    //Copying data to device from host 
    hipMemcpy(a_d, temp_h, sizeof(float) * numvar * (numvar + 1), hipMemcpyHostToDevice);


    //Defining size of Thread Block 
    dim3 dimBlock(numvar + 1, numvar, 1);
    dim3 dimGrid(1, 1, 1);

    //Kernel call 
    Kernel << <dimGrid, dimBlock >> > (a_d, b_d, numvar);

    //Coping data to host from device 
    hipMemcpy(temp1_h, b_d, sizeof(float) * numvar * (numvar + 1), hipMemcpyDeviceToHost);

    //Deallocating memory on the device 
    hipFree(a_d);
    hipFree(b_d);
}