
#include <hip/hip_runtime.h>
__global__ void forwardElimKernel(double *mat, int numvar, int k) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x + k + 1;
    if (idx < numvar) {
        double factor = mat[idx * (numvar + 1) + k] / mat[k * (numvar + 1) + k];
        for (int j = k; j <= numvar; j++) {
            mat[idx * (numvar + 1) + j] -= factor * mat[k * (numvar + 1) + j];
        }
    }
}
