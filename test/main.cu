#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cmath>

// ... [CUDA kernel forwardElimKernel defined here] ...
// cuda kernel for forwardElimKernel define
__global__ void forwardElimKernel(double *mat, int numvar, int k) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > k && i < numvar) {
        double term = mat[i * (numvar + 1) + k] / mat[k * (numvar + 1) + k];
        for (int j = k; j < numvar + 1; j++) {
            mat[i * (numvar + 1) + j] -= term * mat[k * (numvar + 1) + j];
        }
    }
}

void forwardElim(double *mat, int numvar) {
    double *d_mat;
    size_t size = numvar * (numvar + 1) * sizeof(double);
    hipMalloc(&d_mat, size);
    hipMemcpy(d_mat, mat, size, hipMemcpyHostToDevice);

    dim3 block(256);
    dim3 grid((numvar + block.x - 1) / block.x);

    for (int k = 0; k < numvar; k++) {
        forwardElimKernel<<<grid, block>>>(d_mat, numvar, k);
        hipDeviceSynchronize();
    }

    hipMemcpy(mat, d_mat, size, hipMemcpyDeviceToHost);
    hipFree(d_mat);
}

// ... [rest of your host code, including backSub function] ...

// function to calculate the values of the unknowns
void backSub(double mat[N][N+1])
{
    double x[N];  // An array to store solution
 
    /* Start calculating from last equation up to the
       first */
    for (int i = N-1; i >= 0; i--)
    {
        /* start with the RHS of the equation */
        x[i] = mat[i][N];
 
        /* Initialize j to i+1 since matrix is upper
           triangular*/
        for (int j=i+1; j<N; j++)
        {
            /* subtract all the lhs values
             * except the coefficient of the variable
             * whose value is being calculated */
            x[i] -= mat[i][j]*x[j];
        }
 
        /* divide the RHS by the coefficient of the
           unknown being calculated */
        x[i] = x[i]/mat[i][i];
    }
 
    printf("\nSolution for the system:\n");
    for (int i=0; i<N; i++)
        printf("%lf\n", x[i]);
}


int main() {
    double mat[N][N+1] = { /* ... your matrix data ... */ };
    
    // Call forward elimination
    forwardElim((double *)mat, N);

    // Perform back substitution on the host
    backSub(mat);

    return 0;
}
